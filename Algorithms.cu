#include "hip/hip_runtime.h"
#include "Algorithms.h"
#include "Algorithms.cuh"

#include "ExpManager.h"
#include "ThreefryGPU.h"
#include "GPUDna.cuh"

#include <cstdint>
#include <stdio.h>
#include <unistd.h>

#include <iostream>

#include<hip/hip_runtime.h>
#include<hip/hip_runtime_api.h>

using namespace std;

#define DEBUG 1
// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
    if (result != hipSuccess) {
        fprintf(stderr, "CUDA Runtime Error: %s\n",
                hipGetErrorString(result));
        assert(result == hipSuccess);
    }
#endif
    return result;
}


constexpr int32_t PROMOTER_ARRAY_SIZE = 10000;

void transfer_in(ExpManager* exp_m, bool first_gen) {
    exp_m->rng_->initDevice();

  std::vector<size_t> host_dna_size(exp_m->nb_indivs_);
  std::vector<size_t> host_dna_offset(exp_m->nb_indivs_);

  // Compute sizes:
  // * global_dna_size
  // * host_dna_offset[]
  // * host_max_dna_size
  // * host_dna_size[]
  global_dna_size = 0;
  for (int i = 0; i < exp_m->nb_indivs_; i++) {
    host_dna_offset[i] = global_dna_size;
    global_dna_size += exp_m->internal_organisms_[i]->dna_->seq_.size();
    host_max_dna_size =
        host_max_dna_size < exp_m->internal_organisms_[i]->dna_->seq_.size() ?
        exp_m->internal_organisms_[i]->dna_->seq_.size() : host_max_dna_size;
    host_dna_size[i] = exp_m->internal_organisms_[i]->dna_->seq_.size();
  }

  // Create shorthands
  auto seq0 = exp_m->internal_organisms_[0]->dna_->seq_.data();
  auto len0 = exp_m->internal_organisms_[0]->dna_->seq_.size();

    allocated_global_dna_size = global_dna_size*5;

  // Allocate mem for the meta dna

    checkCuda(hipMalloc((void **) &next_gen_dna, allocated_global_dna_size * sizeof(char)));
  checkCuda(hipMalloc((void**) &dna, allocated_global_dna_size * sizeof(char)));
  // Tranfer **the first** indiv's sequence
  checkCuda(hipMemcpy(dna,
                       seq0,
                       len0 * sizeof(char),
                       hipMemcpyHostToDevice));

  // Send dna_size array
  checkCuda(hipMalloc((void**) &dna_size,
                       exp_m->nb_indivs_ * sizeof(size_t)));
  checkCuda(hipMemcpy(dna_size,
                       host_dna_size.data(), exp_m->nb_indivs_ * sizeof(size_t),
                       hipMemcpyHostToDevice));

        checkCuda(hipMalloc((void **) &nb_mut_bp, 1 * sizeof(unsigned long long int)));
        checkCuda(hipMemset(nb_mut_bp, 0, 1 * sizeof(unsigned long long int)));


  // Launch kernel to clone initial genome into the whole pop
  int x_dim_size = (len0 / 128)+1;
  int y_dim_size = exp_m->nb_indivs_;
  dim3 dimGrid(x_dim_size,y_dim_size);

  clone_init_indiv<<<dimGrid,128>>>(dna_size, dna);

  checkCuda(hipMalloc((void**) &dna_term, allocated_global_dna_size * sizeof(int8_t*)));

  checkCuda(hipMalloc((void**) &start_protein,
                       allocated_global_dna_size * sizeof(int8_t*)));


  checkCuda(hipMalloc((void**) &dna_offset,
                       exp_m->nb_indivs_ * sizeof(size_t)));
  checkCuda(hipMemcpy(dna_offset,
                       host_dna_offset.data(),
                       exp_m->nb_indivs_ * sizeof(size_t),
                       hipMemcpyHostToDevice));

  checkCuda(hipMalloc((void**) &next_gen_dna_offset,
                       exp_m->nb_indivs_ * sizeof(size_t)));


  checkCuda(hipMalloc((void**) &next_gen_dna_size,
                       exp_m->nb_indivs_ * sizeof(size_t)));

  checkCuda(hipMalloc((void**) &nb_mutations,
                       (exp_m->nb_indivs_ + 1) * sizeof(int)));
  checkCuda(hipMemset(nb_mutations, 0, (exp_m->nb_indivs_ + 1) * sizeof(int)));

  checkCuda(hipMalloc((void**) &mutations_offset,
                       exp_m->nb_indivs_ * sizeof(int)));
  checkCuda(hipMemset(mutations_offset, 0, exp_m->nb_indivs_ * sizeof(int)));

  checkCuda(hipMalloc((void**) &mutations_idx,
                       exp_m->nb_indivs_ * sizeof(int)));
  checkCuda(hipMemset(mutations_idx, 0, exp_m->nb_indivs_ * sizeof(int)));

  checkCuda(hipMalloc((void**) &dna_mutator_list,
                       exp_m->nb_indivs_ * sizeof(GPUDnaMutator)));

  current_size_tab_mutation = exp_m->nb_indivs_ * 100;
  checkCuda(hipMalloc(&tab_mutation,
                       current_size_tab_mutation * sizeof(TypeMutation)));

  checkCuda(hipMalloc((void**) &rna_idx,
                       (exp_m->nb_indivs_ + 1) * sizeof(int32_t)));
  checkCuda(hipMemset(rna_idx, 0, (exp_m->nb_indivs_ + 1) * sizeof(int32_t)));

  checkCuda(hipMalloc((void**) &rna_offset,
                       exp_m->nb_indivs_ * sizeof(int32_t)));
  checkCuda(hipMemset(rna_offset, 0, exp_m->nb_indivs_ * sizeof(int32_t)));

  checkCuda(hipMalloc((void**) &protein_idx,
                       (exp_m->nb_indivs_ + 1) * sizeof(int32_t)));
  checkCuda(
      hipMemset(protein_idx, 0, (exp_m->nb_indivs_ + 1) * sizeof(int32_t)));

  checkCuda(hipMalloc((void**) &protein_offset,
                       exp_m->nb_indivs_ * sizeof(int32_t)));
  checkCuda(hipMemset(protein_offset, 0, exp_m->nb_indivs_ * sizeof(int32_t)));

  checkCuda(hipMalloc((void**) &next_generation_reproducer,
                       exp_m->nb_indivs_ * sizeof(size_t)));

  checkCuda(hipMalloc((void**) &nb_promoters,
                       (exp_m->nb_indivs_ + 1) * sizeof(int)));
  checkCuda(hipMemset(nb_promoters, 0, (exp_m->nb_indivs_ + 1) * sizeof(int)));

  checkCuda(hipMalloc((void**) &nb_proteins,
                       (exp_m->nb_indivs_ + 1) * sizeof(int)));
  checkCuda(hipMemset(nb_proteins, 0, (exp_m->nb_indivs_ + 1) * sizeof(int)));

  host_phenotype = (double**) malloc(exp_m->nb_indivs_ * sizeof(double*));
  checkCuda(
      hipMalloc((void***) &phenotype, exp_m->nb_indivs_ * sizeof(double*)));


  host_phenotype_activ = (double**) malloc(exp_m->nb_indivs_ * sizeof(double*));
  checkCuda(hipMalloc((void***) &phenotype_activ,
                       exp_m->nb_indivs_ * sizeof(double*)));


  host_phenotype_inhib = (double**) malloc(exp_m->nb_indivs_ * sizeof(double*));
  checkCuda(hipMalloc((void***) &phenotype_inhib,
                       exp_m->nb_indivs_ * sizeof(double*)));

  for (int indiv_id = 0; indiv_id < exp_m->nb_indivs_; indiv_id++) {
    checkCuda(
        hipMalloc((void**) &host_phenotype[indiv_id], 300 * sizeof(double)));
    checkCuda(hipMemset(host_phenotype[indiv_id], 0.0, 300 * sizeof(double)));

    checkCuda(hipMalloc((void**) &host_phenotype_activ[indiv_id],
                         300 * sizeof(double)));
    checkCuda(
        hipMemset(host_phenotype_activ[indiv_id], 0.0, 300 * sizeof(double)));

    checkCuda(hipMalloc((void**) &host_phenotype_inhib[indiv_id],
                         300 * sizeof(double)));
    checkCuda(
        hipMemset(host_phenotype_inhib[indiv_id], 0.0, 300 * sizeof(double)));
  }

  current_size_rna_list = exp_m->nb_indivs_ * 10000;
  checkCuda(hipMalloc(&rna, current_size_rna_list * sizeof(pRNA)));

  current_size_protein_list = exp_m->nb_indivs_ * 1000;
  checkCuda(hipMalloc(&protein, current_size_protein_list * sizeof(pProtein)));

  checkCuda(
      hipMemcpy(phenotype, host_phenotype, exp_m->nb_indivs_ * sizeof(double*),
                 hipMemcpyHostToDevice));

  checkCuda(hipMemcpy(phenotype_activ, host_phenotype_activ,
                       exp_m->nb_indivs_ * sizeof(double*),
                       hipMemcpyHostToDevice));

  checkCuda(hipMemcpy(phenotype_inhib, host_phenotype_inhib,
                       exp_m->nb_indivs_ * sizeof(double*),
                       hipMemcpyHostToDevice));

  checkCuda(hipMalloc((void**) &target,
                       300 * sizeof(double)));

  double target_host[300];
  for (int i = 0; i < 300; i++) {
    target_host[i] = exp_m->target[i];
  }


  checkCuda(hipMemcpy(target,
                       target_host,
                       300 * sizeof(double), hipMemcpyHostToDevice));

  checkCuda(hipMalloc((void**) &metaerror,
                       exp_m->nb_indivs_ * sizeof(double)));


  checkCuda(hipMalloc((void**) &fitness,
                       exp_m->nb_indivs_ * sizeof(double)));


  //printf("GPU Counter %d\n",exp_m->rng_->counters().size());

  checkCuda(hipMalloc((void**) &gpu_counters,
                       exp_m->rng_->counters().size() *
                       sizeof(unsigned long long)));

  checkCuda(hipMemcpy(gpu_counters, exp_m->rng_->counters().data(),
                       exp_m->rng_->counters().size() *
                       sizeof(unsigned long long), hipMemcpyHostToDevice));

}

__global__
// Copy first indiv's dna into all the other indivs' dna
void clone_init_indiv(size_t* dna_size, char* dna) {
  int dna_chunk_idx = blockIdx.x;
  int indiv_id = blockIdx.y;
  if(indiv_id == 0) return; // don't copy indiv 0 onto itself

  int pos = (dna_chunk_idx*128)+threadIdx.x;

  if (pos < dna_size[0]) {
    dna[indiv_id*dna_size[0] + pos] = dna[pos];
  }
}


__global__
void search_start_stop_RNA(size_t* dna_size, char* dna, size_t* dna_offset, int* nb_promoters,
                           int8_t* dna_term, int nb_indivs, int global_dna_size, unsigned long long* nb_mut_bp) {

    int dna_pos_block = blockIdx.x;
    int indiv_id = blockIdx.y;

    int dna_pos = (dna_pos_block*128)+threadIdx.x;

    __shared__ int nb_prom_block;
    if (threadIdx.x == 0) {
        nb_prom_block = 0;
    }
    __syncthreads();

    if (dna_pos < dna_size[indiv_id] && dna_size[indiv_id] >= PROM_SIZE) {
        dna_term[dna_offset[indiv_id]+dna_pos] = 22;
        //atomicAdd(nb_mut_bp,1);

        int prom_dist[22];
        int term_dist[4];

        for (int motif_id = 0; motif_id < 26; motif_id++) {
            if (motif_id < 22) {
                prom_dist[motif_id] =
                        PROM_SEQ[motif_id] ==
                        dna[dna_pos + motif_id >= dna_size[indiv_id] ? dna_offset[indiv_id]+ dna_pos + motif_id - dna_size[indiv_id]
                                                                     : dna_offset[indiv_id]+ dna_pos + motif_id]
                        ? 0
                        : 1;
            } else if (motif_id >= 22) {
                int t_motif_id = motif_id - 22;
                term_dist[t_motif_id] =
                        dna[dna_pos + t_motif_id >= dna_size[indiv_id] ?
                            dna_offset[indiv_id]+dna_pos + t_motif_id - dna_size[indiv_id] :
                            dna_offset[indiv_id]+ dna_pos + t_motif_id] !=
                        dna[dna_pos - t_motif_id + 10 >= dna_size[indiv_id] ?
                            dna_offset[indiv_id]+ dna_pos - t_motif_id + 10 - dna_size[indiv_id] :
                            dna_offset[indiv_id]+ dna_pos - t_motif_id + 10] ? 1 : 0;
            }
        }

        int8_t dist_prom = prom_dist[0] +
                        prom_dist[1] +
                        prom_dist[2] +
                        prom_dist[3] +
                        prom_dist[4] +
                        prom_dist[5] +
                        prom_dist[6] +
                        prom_dist[7] +
                        prom_dist[8] +
                        prom_dist[9] +
                        prom_dist[10] +
                        prom_dist[11] +
                        prom_dist[12] +
                        prom_dist[13] +
                        prom_dist[14] +
                        prom_dist[15] +
                        prom_dist[16] +
                        prom_dist[17] +
                        prom_dist[18] +
                        prom_dist[19] +
                        prom_dist[20] +
                        prom_dist[21];



        dna_term[dna_offset[indiv_id]+dna_pos] = dist_prom;


        if (dist_prom <= 4) {
            int rna_idx = atomicAdd(&nb_prom_block, 1);
        }


        int dist_term = term_dist[0] +
                        term_dist[1] +
                        term_dist[2] +
                        term_dist[3];
        dna_term[dna_offset[indiv_id]+dna_pos] |= dist_term == 4 ? 1<<7 : 0;

    }

    __syncthreads();

    if (threadIdx.x == 0) {
        atomicAdd(nb_promoters+indiv_id,nb_prom_block);
        atomicAdd(nb_promoters+nb_indivs,nb_prom_block);
    }
}



__global__
void compute_RNA_offset(int* nb_promoters, int* rna_offset) {

    const int indiv_id = blockIdx.x;
    __shared__ int grid_rna_offset;

    if (threadIdx.x == 0) {
        grid_rna_offset = 0;
    }
    __syncthreads();

    {
        int local_rna_offset = 0;
        for (int cpt = threadIdx.x; cpt < indiv_id; cpt += blockDim.x) {
            local_rna_offset += nb_promoters[cpt];
        }

        if (local_rna_offset > 0)
            atomicAdd(&grid_rna_offset, local_rna_offset);
    }
    __syncthreads();
    if (threadIdx.x == 0) {
        rna_offset[indiv_id] = grid_rna_offset;
    }
}



__global__
void fill_RNA( int8_t* dna_term, size_t* dna_size, size_t* dna_offset, int* nb_promoters, int* rna_offset, pRNA* rnas,
               int32_t* rna_idx, int nb_indiv) {
    int dna_pos_block = blockIdx.x;
    int indiv_id = blockIdx.y;

    int dna_pos = (dna_pos_block * 128) + threadIdx.x;

    if (dna_pos < dna_size[indiv_id] && dna_size[indiv_id] >= PROM_SIZE) {
        // Masque le bit de poid fort
        int8_t dist = dna_term[dna_offset[indiv_id]+dna_pos] & (0x7F);


        if (dist <= 4) {
            int local_rna_idx = atomicAdd(rna_idx + indiv_id, 1);
            atomicAdd(rna_idx + nb_indiv, 1);

            rnas[rna_offset[indiv_id] + local_rna_idx].begin = dna_pos;
            rnas[rna_offset[indiv_id] + local_rna_idx].dist = dist;
            rnas[rna_offset[indiv_id] + local_rna_idx].transcribed = false;
            rnas[rna_offset[indiv_id] + local_rna_idx].indiv_id = indiv_id;
        }
    }
}

__global__
void compute_RNA( int8_t* dna_term, size_t* dna_size, size_t* dna_offset, pRNA* rnas,  int global_nb_rna) {
	const int globalIdx = blockIdx.x*blockDim.x+threadIdx.x;

    if (globalIdx < global_nb_rna ) {
        int indiv_id = rnas[globalIdx].indiv_id;
        if (dna_size[indiv_id] >= PROM_SIZE) {
        int k = rnas[globalIdx].begin + 22;
        k = k >= dna_size[indiv_id] ? k - dna_size[indiv_id] : k;
        int k_end = k;
        bool found=false;

        do {

            //printf("%d -- %d %ld\n",indiv_id,k,dna_size[indiv_id]);

            if (dna_term[dna_offset[indiv_id]+k] & (1<<7)) {
                int32_t rna_end =
                        k + 10 >= dna_size[indiv_id] ? k + 10 - dna_size[indiv_id] :
                        k +
                        10;

                int32_t rna_length = 0;

                if (rnas[globalIdx].begin > rna_end)
                    rna_length = dna_size[indiv_id] - rnas[globalIdx].begin + rna_end;
                else
                    rna_length = rna_end - rnas[globalIdx].begin;

                if (rna_length < 19) {
                    rnas[globalIdx].begin = 0;
                    rnas[globalIdx].end = 0;
                    rnas[globalIdx].length = 0;
                    rnas[globalIdx].transcribed = false;
                    break;
                }




                rnas[globalIdx].end = rna_end;
                rnas[globalIdx].transcribed = true;
                rnas[globalIdx].length = rna_length;

                if (rnas[globalIdx].end>=dna_size[indiv_id]) {
                    printf("Termin %d %d S %d %ld\n",
                           rnas[globalIdx].begin,
                           rnas[globalIdx].end,indiv_id,dna_size[indiv_id]);
                    //assert(rnas[globalIdx].end<dna_size[indiv_id]);
                }

                found=true;
                break;
            }

            k++;
            k = k >= dna_size[indiv_id] ? k - dna_size[indiv_id] : k;
        } while (k != k_end);
        }
    } else {
        rnas[globalIdx].begin = 0;
        rnas[globalIdx].end = 0;
        rnas[globalIdx].length = 0;
        rnas[globalIdx].transcribed = false;
    }
}

__global__ void display_RNA( pRNA* rna, size_t* dna_size, int32_t global_nb_rna) {
    for(int i = 0; i < global_nb_rna; i++) {
        if (rna[i].transcribed)
            if (rna[i].end>=dna_size[rna[i].indiv_id]) {
                printf("UIIH %d %d S %d -- %ld\n",rna[i].begin,rna[i].end,rna[i].indiv_id,dna_size[rna[i].indiv_id]);
            }
    }

}

__global__
void compute_start_protein(int8_t* start_protein, size_t* dna_size, size_t* dna_offset, pRNA* rna, char* dna, int32_t* nb_proteins,
                           int32_t global_nb_rna, int nb_indiv) {

    const int globalIdx = blockIdx.x*blockDim.x+threadIdx.x;
    int nb_prot = 0;

    if (globalIdx < global_nb_rna) {
        if (rna[globalIdx].transcribed) {
            const int indiv_id = rna[globalIdx].indiv_id;
            int c_pos = rna[globalIdx].begin;
            if (rna[globalIdx].length > 22) {
                c_pos += 22;
                c_pos =
                            c_pos >= dna_size[indiv_id] ? c_pos - dna_size[indiv_id] : c_pos;
                int count_loop=0;

                if (rna[globalIdx].end>=dna_size[indiv_id]) {
                    printf("ator %d S %d\n",rna[globalIdx].end,indiv_id);
                    assert(rna[globalIdx].end<dna_size[indiv_id]);
                }

                while (c_pos != rna[globalIdx].end) {
                    //if (indiv_id==606) printf("%d -- %d %d\n",indiv_id,c_pos,rna[globalIdx].end);
                    bool start = false;
                    int t_pos, k_t;
                    for (int k = 0; k < 9; k++) {
                        k_t = k >= 6 ? k + 4 : k;
                        t_pos = c_pos + k_t >= dna_size[indiv_id] ? c_pos + k_t -
                                                                    dna_size[indiv_id] :
                                c_pos + k_t;
                        count_loop++;
                        if (count_loop>10000) {printf("%d %d %d %d %d %d %d %ld\n",indiv_id,globalIdx,k,
                                                     c_pos,t_pos,
                                                     rna[globalIdx].begin,rna[globalIdx].end,
                                                     dna_size[indiv_id]);assert(0);}
                        if (dna[dna_offset[indiv_id]+t_pos] == SHINE_DAL_SEQ[k]) {
                            start = true;
                        } else {
                            start = false;
                            break;
                        }
                    }


                    start_protein[dna_offset[indiv_id]+c_pos] = start;

                    if (start) {
                        nb_prot++;
                    }

                    c_pos++;
                    c_pos =
                            c_pos >= dna_size[indiv_id] ? c_pos - dna_size[indiv_id] : c_pos;
                }
            }

            atomicAdd(nb_proteins+indiv_id,nb_prot);
            atomicAdd(nb_proteins+nb_indiv,nb_prot);
        }
    }

}


__global__
void compute_protein_offset(int32_t* nb_proteins, int* protein_offset) {

    const int indiv_id = blockIdx.x;
    __shared__ int grid_protein_offset;

    if (threadIdx.x == 0) {
        grid_protein_offset = 0;
    }
    __syncthreads();

    {
        int local_protein_offset = 0;
        for (int cpt = threadIdx.x; cpt < indiv_id; cpt += blockDim.x) {
            local_protein_offset += nb_proteins[cpt];
        }

        if (local_protein_offset > 0)
            atomicAdd(&grid_protein_offset, local_protein_offset);
    }
    __syncthreads();
    if (threadIdx.x == 0) {
        protein_offset[indiv_id] = grid_protein_offset;
    }
}


__global__ void fill_protein(int8_t* start_protein, size_t* dna_offset, int* protein_idx,
                             int* protein_offset, pRNA* rna, pProtein* protein, size_t* dna_size,
                             int32_t global_nb_rna, int nb_indiv) {
	const int globalIdx = blockIdx.x*blockDim.x+threadIdx.x;

    if (globalIdx < global_nb_rna) {
        if (rna[globalIdx].transcribed) {
            int indiv_id = rna[globalIdx].indiv_id;
            int c_pos = rna[globalIdx].begin;
            if (rna[globalIdx].length > 22) {
                c_pos += 22;
                c_pos =
                        c_pos >= dna_size[indiv_id] ? c_pos - dna_size[indiv_id] : c_pos;

                while (c_pos != rna[globalIdx].end) {
                    if (start_protein[dna_offset[indiv_id]+c_pos] == 1) {

                        int local_protein_idx = atomicAdd(protein_idx + indiv_id, 1);
                        atomicAdd(protein_idx + nb_indiv, 1);

                        protein[protein_offset[indiv_id] + local_protein_idx].protein_start = c_pos;
                        protein[protein_offset[indiv_id] + local_protein_idx].indiv_id = rna[globalIdx].indiv_id;
                        protein[protein_offset[indiv_id] + local_protein_idx].stop_RNA = rna[globalIdx].end;
                        protein[protein_offset[indiv_id] + local_protein_idx].translated = false;
                        protein[protein_offset[indiv_id] + local_protein_idx].e = 1.0 -
                                                       fabs(((double) rna[globalIdx].dist)) /
                                                       5.0;
                    }

                    c_pos++;
                    c_pos =
                            c_pos >= dna_size[indiv_id] ? c_pos - dna_size[indiv_id] : c_pos;
                }
            }

        }
    }
}


__global__
void compute_proteins( int8_t* start_protein, size_t* dna_size, size_t* dna_offset, pProtein* protein, char* dna,
                       int32_t global_nb_protein) {
    __shared__ int next_protein_idx;
    if (threadIdx.x == 0) {
        next_protein_idx = 0;
    }
    __syncthreads();

    int local_protein_idx = atomicAdd(&next_protein_idx,1);

    while (local_protein_idx < global_nb_protein) {
        int indiv_id = protein[local_protein_idx].indiv_id;

        int start_protein_pos =  protein[local_protein_idx].protein_start + 13;
        int length = -1;
            start_protein_pos = start_protein_pos >= dna_size[indiv_id] ?
                                start_protein_pos - dna_size[indiv_id]
                                                                        : start_protein_pos;

            if (protein[local_protein_idx].protein_start < protein[local_protein_idx].stop_RNA) {
                length = protein[local_protein_idx].stop_RNA - protein[local_protein_idx].protein_start;
            } else {
                length = dna_size[indiv_id] - protein[local_protein_idx].protein_start + protein[local_protein_idx].stop_RNA + 1;
            }

            length -= 13;


        bool is_protein = false;
        length+=1;
        length = length - (length%3);

        for (int loop_i = 0; length - loop_i >= 2; loop_i+=3) {
            int t_k;

            start_protein_pos = start_protein_pos >= dna_size[indiv_id] ?
                                start_protein_pos - dna_size[indiv_id]
                                                                        : start_protein_pos;
            is_protein = false;

            for (int k = 0; k < 3; k++) {
                t_k = start_protein_pos + k >= dna_size[indiv_id] ?
                      start_protein_pos - dna_size[indiv_id] + k :
                      start_protein_pos + k;

                if (dna[dna_offset[indiv_id]+t_k] == PROTEIN_END[k]) {
                    is_protein = true;
                } else {
                    is_protein = false;
                    break;
                }
            }

            if (is_protein) {
                int prot_length = -1;
                if (protein[local_protein_idx].protein_start + 13 < t_k) {
                    prot_length = t_k - (protein[local_protein_idx].protein_start + 13);
                } else {
                    prot_length = dna_size[indiv_id] - (protein[local_protein_idx].protein_start + 13) + t_k;
                }

                if (prot_length >= 3) {
                    protein[local_protein_idx].protein_end = t_k;
                    protein[local_protein_idx].protein_length = prot_length;
                    protein[local_protein_idx].translated = true;
                }
                break;
            }


            start_protein_pos += 3;
            start_protein_pos = start_protein_pos >= dna_size[indiv_id] ?
                                start_protein_pos - dna_size[indiv_id]
                                                                        : start_protein_pos;

        }



        local_protein_idx = atomicAdd(&next_protein_idx,1);
    }
}


__global__
void translate_proteins( pProtein* protein, size_t* dna_size, char* dna,  size_t* dna_offset, int32_t global_nb_protein, double w_max) {
    __shared__ int next_protein_idx;
    if (threadIdx.x == 0) {
        next_protein_idx = 0;
    }
    __syncthreads();

    int local_protein_idx = atomicAdd(&next_protein_idx,1);

    while (local_protein_idx < global_nb_protein) {
        int indiv_id = protein[local_protein_idx].indiv_id;

        if (protein[local_protein_idx].translated) {

            int c_pos = protein[local_protein_idx].protein_start, t_pos;
            int end_pos = protein[local_protein_idx].protein_end;
            c_pos += 13;
            end_pos -= 3;

            c_pos = c_pos >= dna_size[indiv_id] ? c_pos - dna_size[indiv_id] : c_pos;
            end_pos = end_pos < 0 ? dna_size[indiv_id] + end_pos : end_pos;

            int8_t value = 0;
            int8_t codon_list[64] = {};
            int8_t codon_idx = 0;
            int32_t count_loop = 0;

            bool contin = true;


            while (count_loop < protein[local_protein_idx].protein_length / 3 && codon_idx < 64) {
                value = 0;
                for (int8_t i = 0; i < 3; i++) {
                    t_pos = c_pos + i >= dna_size[indiv_id] ? c_pos + i - dna_size[indiv_id] : c_pos + i;
                    if (dna[dna_offset[indiv_id]+t_pos] == '1') value += 1 << (CODON_SIZE - i - 1);
                }
                codon_list[codon_idx] = value;
                codon_idx++;

                count_loop++;
                c_pos += 3;
                c_pos = c_pos >= dna_size[indiv_id] ? c_pos - dna_size[indiv_id] : c_pos;
            }


            double M = 0.0;
            double W = 0.0;
            double H = 0.0;

            int32_t nb_m = 0;
            int32_t nb_w = 0;
            int32_t nb_h = 0;

            bool bin_m = false; // Initializing to false will yield a conservation of the high weight bit
            bool bin_w = false; // when applying the XOR operator for the Gray to standard conversion
            bool bin_h = false;


            for (int i = 0; i < codon_idx; i++) {
                switch (codon_list[i]) {
                    case CODON_M0 : {
                        // M codon found
                        nb_m++;

                        // Convert Gray code to "standard" binary code
                        bin_m ^= false; // as bin_m was initialized to false, the XOR will have no effect on the high weight bit

                        // A lower-than-the-previous-lowest weight bit was found, make a left bitwise shift
                        //~ M <<= 1;
                        M *= 2;

                        // Add this nucleotide's contribution to M
                        if (bin_m) M += 1;

                        break;
                    }
                    case CODON_M1 : {
                        // M codon found
                        nb_m++;

                        // Convert Gray code to "standard" binary code
                        bin_m ^= true; // as bin_m was initialized to false, the XOR will have no effect on the high weight bit

                        // A lower-than-the-previous-lowest bit was found, make a left bitwise shift
                        //~ M <<= 1;
                        M *= 2;

                        // Add this nucleotide's contribution to M
                        if (bin_m) M += 1;

                        break;
                    }
                    case CODON_W0 : {
                        // W codon found
                        nb_w++;

                        // Convert Gray code to "standard" binary code
                        bin_w ^= false; // as bin_m was initialized to false, the XOR will have no effect on the high weight bit

                        // A lower-than-the-previous-lowest weight bit was found, make a left bitwise shift
                        //~ W <<= 1;
                        W *= 2;

                        // Add this nucleotide's contribution to W
                        if (bin_w) W += 1;

                        break;
                    }
                    case CODON_W1 : {
                        // W codon found
                        nb_w++;

                        // Convert Gray code to "standard" binary code
                        bin_w ^= true; // as bin_m was initialized to false, the XOR will have no effect on the high weight bit

                        // A lower-than-the-previous-lowest weight bit was found, make a left bitwise shift
                        //~ W <<= 1;
                        W *= 2;

                        // Add this nucleotide's contribution to W
                        if (bin_w) W += 1;

                        break;
                    }
                    case CODON_H0 :
                    case CODON_START : // Start codon codes for the same amino-acid as H0 codon
                    {
                        // H codon found
                        nb_h++;

                        // Convert Gray code to "standard" binary code
                        bin_h ^= false; // as bin_m was initialized to false, the XOR will have no effect on the high weight bit

                        // A lower-than-the-previous-lowest weight bit was found, make a left bitwise shift
                        //~ H <<= 1;
                        H *= 2;

                        // Add this nucleotide's contribution to H
                        if (bin_h) H += 1;

                        break;
                    }
                    case CODON_H1 : {
                        // H codon found
                        nb_h++;

                        // Convert Gray code to "standard" binary code
                        bin_h ^= true; // as bin_m was initialized to false, the XOR will have no effect on the high weight bit

                        // A lower-than-the-previous-lowest weight bit was found, make a left bitwise shift
                        //~ H <<= 1;
                        H *= 2;

                        // Add this nucleotide's contribution to H
                        if (bin_h) H += 1;

                        break;
                    }
                }
            }



            //  ----------------------------------------------------------------------------------
            //  2) Normalize M, W and H values in [0;1] according to number of codons of each kind
            //  ----------------------------------------------------------------------------------
            protein[local_protein_idx].m = nb_m != 0 ? M / (pow(2, nb_m) - 1) : 0.5;
            protein[local_protein_idx].w = nb_w != 0 ? W / (pow(2, nb_w) - 1) : 0.0;
            protein[local_protein_idx].h = nb_h != 0 ? H / (pow(2, nb_h) - 1) : 0.5;

            //  ------------------------------------------------------------------------------------
            //  3) Normalize M, W and H values according to the allowed ranges (defined in macros.h)
            //  ------------------------------------------------------------------------------------
            // x_min <= M <= x_max
            // w_min <= W <= w_max
            // h_min <= H <= h_max
            protein[local_protein_idx].m = (X_MAX - X_MIN) * protein[local_protein_idx].m + X_MIN;
            protein[local_protein_idx].w = (w_max - W_MIN) * protein[local_protein_idx].w + W_MIN;
            protein[local_protein_idx].h = (H_MAX - H_MIN) * protein[local_protein_idx].h + H_MIN;

            if (nb_m == 0 || nb_w == 0 || nb_h == 0 || protein[local_protein_idx].w == 0.0 ||
                protein[local_protein_idx].h == 0.0) {
                protein[local_protein_idx].is_functional = false;
            } else {
                protein[local_protein_idx].is_functional = true;
            }

        }

        local_protein_idx = atomicAdd(&next_protein_idx,1);
    }
}


__global__
void compute_phenotype( pProtein* protein, int32_t global_nb_protein, double** phenotype,
                        double** phenotype_activ, double** phenotype_inhib, int nb_indiv) {
    __shared__ int next_protein_idx;

    if (threadIdx.x == 0) {
        next_protein_idx = 0;
    }
    __syncthreads();

    int local_protein_idx = atomicAdd(&next_protein_idx,1);

    while (local_protein_idx < global_nb_protein) {

        int indiv_id = protein[local_protein_idx].indiv_id;

        if (protein[local_protein_idx].translated) {
            if (fabs(protein[local_protein_idx].w) < 1e-15 ||
                fabs(protein[local_protein_idx].h) < 1e-15) {

            } else {
                if (protein[local_protein_idx].is_functional) {

                    // Compute triangle points' coordinates
                    double x0 = protein[local_protein_idx].m -
                                protein[local_protein_idx].w;
                    double x1 = protein[local_protein_idx].m;
                    double x2 = protein[local_protein_idx].m +
                                protein[local_protein_idx].w;

                    int ix0 = (int) (x0 * 300);
                    int ix1 = (int) (x1 * 300);
                    int ix2 = (int) (x2 * 300);

                    if (ix0 < 0) ix0 = 0; else if (ix0 > (299)) ix0 = 299;
                    if (ix1 < 0) ix1 = 0; else if (ix1 > (299)) ix1 = 299;
                    if (ix2 < 0) ix2 = 0; else if (ix2 > (299)) ix2 = 299;

                    // Compute the first equation of the triangle
                    double incY = (protein[local_protein_idx].h *
                                   protein[local_protein_idx].e) / (ix1 - ix0);
                    int count = 1;

                    // Updating value between x0 and x1
                    for (int i = ix0 + 1; i < ix1; i++) {
                        if (protein[local_protein_idx].h > 0)
                            atomicAdd(&phenotype_activ[indiv_id][i], (incY * (count++)));
                        else
                            atomicAdd(&phenotype_inhib[indiv_id][i], (incY * (count++)));

                    }


                    if (protein[local_protein_idx].h > 0) {
                        atomicAdd(&phenotype_activ[indiv_id][ix1],
                                  (protein[local_protein_idx].h *
                                   protein[local_protein_idx].e));
                    } else
                        atomicAdd(&phenotype_inhib[indiv_id][ix1],
                                  (protein[local_protein_idx].h *
                                   protein[local_protein_idx].e));


                    // Compute the second equation of the triangle
                    incY =
                            (protein[local_protein_idx].h *
                             protein[local_protein_idx].e) /
                            (ix2 - ix1);
                    count = 1;

                    // Updating value between x1 and x2
                    for (int i = ix1 + 1; i < ix2; i++) {
                        if (protein[local_protein_idx].h > 0)
                            atomicAdd(&phenotype_activ[indiv_id][i],
                                      ((protein[local_protein_idx].h *
                                        protein[local_protein_idx].e) -
                                       (incY * (count++))));
                        else
                            atomicAdd(&phenotype_inhib[indiv_id][i],
                                      ((protein[local_protein_idx].h *
                                        protein[local_protein_idx].e) -
                                       (incY * (count++))));
                    }

                }
            }
        }

        local_protein_idx = atomicAdd(&next_protein_idx,1);
    }

    __syncthreads();

}


__global__ void compute_metaerror_fitness(double selection_pressure,double** phenotype,
                                          double** phenotype_activ,double** phenotype_inhib,
                                          double* target,
                                          double* metaerror, double* fitness) {
    int indiv_id = blockIdx.x;

    int fuzzy_idx = threadIdx.x;

        if (phenotype_activ[indiv_id][fuzzy_idx] > 1.0)
            phenotype_activ[indiv_id][fuzzy_idx] = 1.0;
        if (phenotype_inhib[indiv_id][fuzzy_idx] < -1.0)
            phenotype_inhib[indiv_id][fuzzy_idx] = -1.0;

        phenotype[indiv_id][fuzzy_idx] = phenotype_activ[indiv_id][fuzzy_idx] +
                                                   phenotype_inhib[indiv_id][fuzzy_idx];

    __shared__ double delta[300];

    if (phenotype[indiv_id][fuzzy_idx] > 1) phenotype[indiv_id][fuzzy_idx] = 1;
    if (phenotype[indiv_id][fuzzy_idx] < 0) phenotype[indiv_id][fuzzy_idx] = 0;

    delta[fuzzy_idx] = phenotype[indiv_id][fuzzy_idx] - target[fuzzy_idx];

    __syncthreads();

    if (threadIdx.x == 0) {
        metaerror[indiv_id] = 0;

        for (int i = 0; i < 299; i++) {
            metaerror[indiv_id] +=
                    ((fabs(delta[i]) +
                      fabs(delta[i + 1])) / (600.0));
        }

        fitness[indiv_id] = exp(
                -selection_pressure * ((double)metaerror[indiv_id]));
    }
}


__device__ int32_t Threefry::Device::roulette_random(double* probs, int32_t nb_elts)
{
    double pick_one = 0.0;

    while (pick_one == 0.0)
    {
        pick_one = randomDouble();
    }

    int32_t found_org = 0;

    pick_one -= probs[0];
    while (pick_one > 0)
    {
        assert(found_org<nb_elts-1);

        pick_one -= probs[++found_org];
    }
    return found_org;
}

__global__ void selection(double* fitness, int* next_generation_reproducer, unsigned long long* gpu_counters,
                          int grid_width, int grid_height, int nb_indiv) {
    int indiv_id = blockIdx.x;
    int neightbor = threadIdx.x;

    __shared__ double local_fit_array[NEIGHBORHOOD_SIZE];
    __shared__ double probs[NEIGHBORHOOD_SIZE];
    __shared__ int   count;
    __shared__ double    sum_local_fit;

    int32_t x = indiv_id / grid_height;
    int32_t y = indiv_id % grid_height;

    int cur_x,cur_y;

    if (threadIdx.x == 0) {
        count             = 0;
        sum_local_fit     = 0.0;
    }

    __syncthreads();

    if (threadIdx.x == 0) {

        for (int8_t i = -1; i < SELECTION_SCOPE_X - 1; i++) {
            for (int8_t j = -1; j < SELECTION_SCOPE_Y - 1; j++) {
                cur_x = (x + i + grid_width) % grid_width;
                cur_y = (y + j + grid_height) % grid_height;

                local_fit_array[count] = fitness[cur_x * grid_height + cur_y];
                atomicAdd(&sum_local_fit, local_fit_array[count]);

                count++;
            }
        }
    }

    __syncthreads();


    //for(int16_t i = 0 ; i < NEIGHBORHOOD_SIZE ; i++) {

        probs[neightbor] = local_fit_array[neightbor]/sum_local_fit;

    __syncthreads();

    if (threadIdx.x == 0) {
        Threefry::Device rng(gpu_counters,indiv_id,Threefry::Phase::REPROD,nb_indiv);
        int found_org = rng.roulette_random(probs, NEIGHBORHOOD_SIZE);

        int x_offset = (found_org / SELECTION_SCOPE_X) - 1;
        int y_offset = (found_org % SELECTION_SCOPE_Y) - 1;

        next_generation_reproducer[indiv_id] = ((x + x_offset + grid_width) % grid_width) * grid_height +
                                               ((y + y_offset + grid_height) % grid_height);
    }
}


__constant__ double cof[6] = {  76.18009172947146,
                                -86.50532032941677,
                                24.01409824083091,
                                -1.231739572450155,
                                0.1208650973866179e-2,
                                -0.5395239384953e-5 };



// Returns the value ln[gamma(X)] for X.
// The gamma function is defined by the integral  gamma(z) = int(0, +inf, t^(z-1).e^(-t)dt).
// When the argument z is an integer, the gamma function is just the familiar factorial
// function, but offset by one, n! = gamma(n + 1).
__device__ static double gammln(double X)
{
    double x, y, tmp, ser;

    y = x = X;
    tmp = x + 5.5;
    tmp -= (x+0.5) * log(tmp);
    ser = 1.000000000190015;

    for (int8_t j = 0 ; j <= 5 ; j++)
    {
        ser += cof[j] / ++y;
    }

    return -tmp + log(2.5066282746310005 * ser / x);
}


__device__ 
int32_t Threefry::Device::binomial_random(int32_t nb_drawings, double prob)
{
    int32_t nb_success;

    // The binomial distribution is invariant under changing
    // ProbSuccess to 1-ProbSuccess, if we also change the answer to
    // NbTrials minus itself; we ll remember to do this below.
    double p;
    if (prob <= 0.5) p = prob;
    else p = 1.0 - prob;

    // mean of the deviate to be produced
    double mean = nb_drawings * p;


    if (nb_drawings < 25)
        // Use the direct method while NbTrials is not too large.
        // This can require up to 25 calls to the uniform random.
    {
        nb_success = 0;
        for (int32_t j = 1 ; j <= nb_drawings ; j++)
        {
            if (randomDouble() < p) nb_success++;
        }
    }
    else if (mean < 1.0)
        // If fewer than one event is expected out of 25 or more trials,
        // then the distribution is quite accurately Poisson. Use direct Poisson method.
    {
        double g = exp(-mean);
        double t = 1.0;
        int32_t j;
        for (j = 0; j <= nb_drawings ; j++)
        {
            t = t * randomDouble();
            if (t < g) break;
        }

        if (j <= nb_drawings) nb_success = j;
        else nb_success = nb_drawings;
    }

    else
        // Use the rejection method.
    {
        double en     = nb_drawings;
        double oldg   = gammln(en + 1.0);
        double pc     = 1.0 - p;
        double plog   = log(p);
        double pclog  = log(pc);

        // rejection method with a Lorentzian comparison function.
        double sq = sqrt(2.0 * mean * pc);
        double angle, y, em, t;
        do
        {
            do
            {
                angle = M_PI * randomDouble();
                y = tan(angle);
                em = sq*y + mean;
            } while (em < 0.0 || em >= (en + 1.0)); // Reject.

            em = floor(em); // Trick for integer-valued distribution.
            t = 1.2 * sq * (1.0 + y*y)
                * exp(oldg - gammln(em + 1.0) - gammln(en - em + 1.0) + em * plog + (en - em) * pclog);

        } while (randomDouble() > t); // Reject. This happens about 1.5 times per deviate, on average.

        nb_success = (int32_t) rint(em);
    }


    // Undo the symmetry transformation.
    if (p != prob) nb_success = nb_drawings - nb_success;

    return nb_success;
}


__global__
void generate_mutations(unsigned long long* gpu_counters, size_t* dna_size, int* nb_mutations,
                        GPUDnaMutator* dna_mutator_list,int* next_generation_reproducer,
                        int nb_indivs, double mutation_rate) {
    int indiv_id = blockIdx.x;

    Threefry::Device rng(gpu_counters,indiv_id,Threefry::Phase::MUTATION,nb_indivs);

    double mutation_r = mutation_rate;
    int prev_gen_id = next_generation_reproducer[indiv_id];
    size_t prev_gen_size = dna_size[prev_gen_id];

    // Small mutations
    dna_mutator_list[indiv_id].nb_swi_ = rng.
            binomial_random(prev_gen_size, mutation_r);
    dna_mutator_list[indiv_id].nb_mut_ = dna_mutator_list[indiv_id].nb_swi_;
    dna_mutator_list[indiv_id].cpt_mut_ = dna_mutator_list[indiv_id].nb_mut_;

    nb_mutations[indiv_id] = dna_mutator_list[indiv_id].nb_mut_;
    atomicAdd(nb_mutations+nb_indivs,nb_mutations[indiv_id]);
}

__global__
void compute_tab_mutations_offset(int* nb_mutations, int* mutations_offset) {
    const int indiv_id = blockIdx.x;
    __shared__ int grid_mutation_offset;

    if (threadIdx.x == 0) {
        grid_mutation_offset = 0;
    }
    __syncthreads();

    {
        int local_mutation_offset = 0;
        for (int cpt = threadIdx.x; cpt < indiv_id; cpt += blockDim.x) {
            local_mutation_offset += nb_mutations[cpt];
        }

        if (local_mutation_offset > 0)
            atomicAdd(&grid_mutation_offset, local_mutation_offset);
    }
    __syncthreads();
    if (threadIdx.x == 0) {
        mutations_offset[indiv_id] = grid_mutation_offset;
    }
}

__device__ static int mod(int a, int b)
{

    assert(b > 0);

    while (a < 0)  a += b;
    while (a >= b) a -= b;

    return a;
}

__global__
void predict_size_v2(size_t* dna_size, size_t* next_gen_dna_size, GPUDnaMutator* dna_mutator_list,
                     TypeMutation* tab_mut,
                     int* nb_mutations, int* mutations_offset,
                     unsigned long long* gpu_counters,int* next_generation_reproducer,
                     int max_genome_length,
                     int min_genome_length, int nb_indiv) {
    const int indiv_id = blockIdx.x;

    int random_value;

    int transient_size = dna_size[next_generation_reproducer[indiv_id]];

    Threefry::Device rng(gpu_counters,indiv_id,Threefry::Phase::MUTATION,nb_indiv);

    for (int mut_idx = 0; mut_idx <  dna_mutator_list[indiv_id].nb_mut_; mut_idx++) {
            dna_mutator_list[indiv_id].cpt_mut_--;

                dna_mutator_list[indiv_id].nb_swi_--;

                int pos = rng.random(transient_size);

                tab_mut[mutations_offset[indiv_id]+mut_idx].type_ = MutationEventType::DO_SWITCH;
                tab_mut[mutations_offset[indiv_id]+mut_idx].pos_1_ = pos;

    }

    next_gen_dna_size[indiv_id] = transient_size;
}


__global__ void display_mut(TypeMutation* tab_mut,
                            int* nb_mutations, int* mutations_offset) {
    for (int indiv_id = 0; indiv_id < 25; indiv_id++) {
        printf("nb mut %d : %d %d\n",indiv_id,nb_mutations[indiv_id],mutations_offset[indiv_id]);
        for (int i = 0; i < nb_mutations[indiv_id]; i++) {
            printf("%d -- %d %d %d %d %d %c%c%c%c%c%c %d\n", i, tab_mut[mutations_offset[indiv_id] + i].type_,
                   tab_mut[mutations_offset[indiv_id] + i].pos_1_,
                   tab_mut[mutations_offset[indiv_id] + i].pos_2_,
                   tab_mut[mutations_offset[indiv_id] + i].pos_3_,
                   tab_mut[mutations_offset[indiv_id] + i].number_,
                   tab_mut[mutations_offset[indiv_id] + i].seq[0],
                   tab_mut[mutations_offset[indiv_id] + i].seq[1],
                   tab_mut[mutations_offset[indiv_id] + i].seq[2],
                   tab_mut[mutations_offset[indiv_id] + i].seq[3],
                   tab_mut[mutations_offset[indiv_id] + i].seq[4],
                   tab_mut[mutations_offset[indiv_id] + i].seq[5],
                   tab_mut[mutations_offset[indiv_id] + i].transient_size);
        }
    }

}


__global__
void compute_next_gen_dna_offset(size_t* next_gen_dna_size, size_t* next_gen_dna_offset) {

    const int indiv_id = blockIdx.x;
    __shared__ int grid_dna_offset;

    if (threadIdx.x == 0) {
        grid_dna_offset = 0;
    }
    __syncthreads();

    {
        int local_dna_offset = 0;
        for (int cpt = threadIdx.x; cpt < indiv_id; cpt += blockDim.x) {
            local_dna_offset += next_gen_dna_size[cpt];
        }

        if (local_dna_offset > 0)
            atomicAdd(&grid_dna_offset, local_dna_offset);
    }
    __syncthreads();
    if (threadIdx.x == 0) {
        next_gen_dna_offset[indiv_id] = grid_dna_offset;
    }
}

__global__ void do_mutation_v2(TypeMutation* tab_mut,
                               int* nb_mutations, size_t* dna_size, size_t* dna_offset, char* dna,
                               char* next_gen_dna, size_t* next_gen_dna_size, size_t* next_gen_dna_offset,
                               int* next_generation_reproducer,  int* mutations_offset, unsigned long long int* nb_mut_bp) {

    int dna_pos_block = blockIdx.x;
    int indiv_id = blockIdx.y;
    int32_t locus = (dna_pos_block*128)+threadIdx.x;
    int32_t next_locus = locus;

    if (locus < next_gen_dna_size[indiv_id]) {

        int8_t mutate = 0;
        int nb_events = nb_mutations[indiv_id];


        for (; nb_events > 0; nb_events--) {
            auto &mut = tab_mut[mutations_offset[indiv_id]+nb_events - 1];

            switch (mut.type_) {
                case DO_SWITCH:
                    if (locus == mut.pos_1_)
                        mutate = not mutate;
                    break;
            }
        }

        assert(locus >= 0);

        assert(locus < dna_size[next_generation_reproducer[indiv_id]]);

        auto base = dna[dna_offset[next_generation_reproducer[indiv_id]]+locus];
        if (mutate) base = (base == '0') ? '1' : '0';

        next_gen_dna[next_gen_dna_offset[indiv_id]+next_locus] = base;

    }
}


void run_a_step_on_GPU(int nb_indiv, double w_max, double selection_pressure, int grid_width, int grid_height, double mutation_rate) {
    int x_dim_size = (host_max_dna_size / 128)+1;

    int y_dim_size = nb_indiv;

    dim3 dimGrid(x_dim_size,y_dim_size);

    search_start_stop_RNA<<<dimGrid,128>>>(dna_size,dna,dna_offset,
            nb_promoters,dna_term,nb_indiv,global_dna_size,nb_mut_bp);

    int total_nb_promoters_host;
    checkCuda(hipMemcpy(&total_nb_promoters_host,
                         nb_promoters+nb_indiv, sizeof(int), hipMemcpyDeviceToHost));

    if (total_nb_promoters_host > current_size_rna_list) {
        checkCuda(hipFree(rna));
        current_size_rna_list = total_nb_promoters_host * 1.1;
        checkCuda(hipMalloc(&rna,current_size_rna_list* sizeof(pRNA)));
    }

    compute_RNA_offset<<<nb_indiv,128>>>(nb_promoters,rna_offset);

    fill_RNA<<<dimGrid,128>>>( dna_term, dna_size,dna_offset, nb_promoters, rna_offset, rna, rna_idx,nb_indiv);

    int global_nb_rna;
    checkCuda(hipMemcpy(&global_nb_rna,
                         rna_idx+nb_indiv, sizeof(int), hipMemcpyDeviceToHost));


    compute_RNA<<<global_nb_rna/128+1,128>>>( dna_term,dna_size, dna_offset, rna, global_nb_rna);

    hipDeviceSynchronize();
    compute_start_protein<<<global_nb_rna,1>>>(start_protein, dna_size, dna_offset, rna, dna, nb_proteins,
            global_nb_rna, nb_indiv);
    hipDeviceSynchronize();

        int total_nb_protein_host;
    checkCuda(hipMemcpy(&total_nb_protein_host,
                         nb_proteins+nb_indiv, sizeof(int), hipMemcpyDeviceToHost));

    if (total_nb_protein_host > current_size_protein_list) {
        checkCuda(hipFree(protein));
        current_size_protein_list = total_nb_protein_host * 1.1;
        checkCuda(hipMalloc(&protein,current_size_protein_list* sizeof(pProtein)));
    }

    compute_protein_offset<<<nb_indiv,128>>>(nb_proteins, protein_offset);

    fill_protein<<<global_nb_rna/128+1,128>>>(start_protein,dna_offset, protein_idx, protein_offset, rna, protein,
            dna_size, global_nb_rna, nb_indiv);

    int global_nb_protein;
    checkCuda(hipMemcpy(&global_nb_protein,
                         protein_idx+nb_indiv, sizeof(int), hipMemcpyDeviceToHost));

    compute_proteins<<<1,128>>>( start_protein, dna_size, dna_offset,protein, dna, global_nb_protein);

    translate_proteins<<<1,128>>>( protein, dna_size, dna, dna_offset, global_nb_protein, w_max);

    compute_phenotype<<<1,128>>>( protein,global_nb_protein, phenotype,
            phenotype_activ,phenotype_inhib, nb_indiv);

    compute_metaerror_fitness<<<nb_indiv,300>>>(selection_pressure,phenotype,
                                    phenotype_activ,phenotype_inhib,
                                   target,
                                   metaerror, fitness);

    // SELECTION
    selection<<<nb_indiv,NEIGHBORHOOD_SIZE>>>(fitness,next_generation_reproducer,gpu_counters,
            grid_width,grid_height,nb_indiv);

    // GENERATE MUTATION + PREDICT
    generate_mutations<<<nb_indiv,1>>>(gpu_counters,dna_size,nb_mutations,dna_mutator_list,
            next_generation_reproducer,
            nb_indiv,mutation_rate);


    compute_tab_mutations_offset<<<nb_indiv,1>>>(nb_mutations,mutations_offset);

    int total_nb_mutations_host;
    checkCuda(hipMemcpy(&total_nb_mutations_host,
                         nb_mutations+nb_indiv, sizeof(int), hipMemcpyDeviceToHost));

    if (total_nb_mutations_host > current_size_tab_mutation) {
        checkCuda(hipFree(tab_mutation));
        current_size_tab_mutation = total_nb_mutations_host * 1.1;
        checkCuda(hipMalloc(&tab_mutation,current_size_tab_mutation* sizeof(TypeMutation)));
    }

    int min_genome_length_  = 10;
    int max_genome_length_  = 10000000;

    predict_size_v2<<<nb_indiv,1>>>(dna_size, next_gen_dna_size, dna_mutator_list,
            tab_mutation,nb_mutations,mutations_offset,gpu_counters,next_generation_reproducer,
            max_genome_length_,min_genome_length_,nb_indiv);
    hipDeviceSynchronize();
    // DO MUTATION

    std::vector <size_t> host_dna_size(
            nb_indiv);

    checkCuda(hipMemcpy(host_dna_size.data(),
                         next_gen_dna_size, nb_indiv * sizeof(size_t), hipMemcpyDeviceToHost));

    global_dna_size=0;
    for (int i = 0; i < nb_indiv; i++) {
        global_dna_size += host_dna_size[i];
        host_max_dna_size = host_max_dna_size < host_dna_size[i] ?
                            host_dna_size[i] : host_max_dna_size;
    }

    bool haveChange = false;
    if (global_dna_size >= allocated_global_dna_size) {
        haveChange = true;
        allocated_global_dna_size = global_dna_size*2;

        checkCuda(hipMalloc((void **) &next_gen_dna, allocated_global_dna_size * sizeof(char)));
        checkCuda(hipFree(dna_term));
        checkCuda(hipMalloc((void **) &dna_term, allocated_global_dna_size * sizeof(int8_t * )));

        checkCuda(hipFree(start_protein));
        checkCuda(hipMalloc((void **) &start_protein, allocated_global_dna_size * sizeof(int8_t * )));
    }



    compute_next_gen_dna_offset<<<nb_indiv,128>>>(next_gen_dna_size, next_gen_dna_offset);

    x_dim_size = (host_max_dna_size / 128)+1;
    y_dim_size = nb_indiv;

    dim3 dimGrid2(x_dim_size,y_dim_size);


    do_mutation_v2<<<dimGrid2,128>>>(tab_mutation,
                                   nb_mutations, dna_size, dna_offset, dna,
                                   next_gen_dna, next_gen_dna_size, next_gen_dna_offset,next_generation_reproducer,
            mutations_offset,nb_mut_bp);

    //printf("DNA 1 %p\n",dna);
    //next_generation_dna_read<<<1,1>>>(next_gen_dna, next_gen_dna_offset,next_gen_dna_size, global_dna_size);

    // SWITCH STRUCTURE

    int block = ceil(nb_indiv/32);
    do_memset<<<block,32>>>(phenotype_activ,phenotype_inhib,nb_mutations,rna_idx,protein_idx,nb_proteins,
            nb_promoters,next_gen_dna_size,
            nb_indiv);

    //allocate_next_gen(nb_indiv);
    //printf("DNA 2 %p\n",dna);

    size_t* tmp_dna_size = dna_size;
    dna_size = next_gen_dna_size;
    next_gen_dna_size = tmp_dna_size;


    size_t* tmp_dna_offset = dna_offset;
    dna_offset = next_gen_dna_offset;
    next_gen_dna_offset = tmp_dna_offset;

    //global_dna_size = global_next_gen_dna_size;
    hipDeviceSynchronize();

    assert(dna!=0);
    //printf("DNA 3 %p\n",dna);

    if (haveChange) {
        checkCuda(hipFree(dna));
        checkCuda(hipMalloc((void **) &dna, allocated_global_dna_size * sizeof(char)));
    }


    //printf("DNA 4 %p\n",dna);

    hipDeviceSynchronize();


    char* dna_tmp = dna;
    dna = next_gen_dna;
    next_gen_dna = dna_tmp;

    //  clean(exp_m);
}

void allocate_next_gen(int nb_indiv) {
    for (int indiv_id = 0; indiv_id < nb_indiv; indiv_id++) {
        checkCuda(hipMemset(host_phenotype[indiv_id], 0.0, 300 * sizeof(double)));
        checkCuda(hipMemset(host_phenotype_activ[indiv_id], 0.0, 300 * sizeof(double)));
        checkCuda(hipMemset(host_phenotype_inhib[indiv_id], 0.0, 300 * sizeof(double)));
    }

    checkCuda(hipMemset(nb_mutations, 0, (nb_indiv+1) * sizeof(int)));
    checkCuda(hipMemset(mutations_offset, 0, nb_indiv * sizeof(int)));
    checkCuda(hipMemset(mutations_idx, 0, nb_indiv * sizeof(int)));

    checkCuda(hipMemset(rna_idx, 0, (nb_indiv+1) * sizeof(int32_t)));
    checkCuda(hipMemset(rna_offset, 0, nb_indiv * sizeof(int32_t)));

    checkCuda(hipMemset(protein_idx, 0, (nb_indiv+1) * sizeof(int32_t)));
    checkCuda(hipMemset(protein_offset, 0, nb_indiv * sizeof(int32_t)));
    checkCuda(hipMemset(nb_proteins, 0, (nb_indiv+1) * sizeof(int)));

    checkCuda(hipMemset(nb_promoters, 0, (nb_indiv+1) * sizeof(int)));
}

__global__
void do_memset(double** phenotype_activ, double** phenotype_inhib, int* nb_mutations, int32_t* rna_idx,
               int32_t* protein_idx, int* nb_proteins, int* nb_promoters,
               size_t* dna_size,
               int nb_indiv) {
    const int indiv_id = blockIdx.x * blockDim.x + threadIdx.x;

    if (indiv_id < nb_indiv) {
        for (int i = 0; i < 300; i++) {
            phenotype_inhib[indiv_id][i] = 0;
            phenotype_activ[indiv_id][i] = 0;
        }

        rna_idx[indiv_id] = 0;
        protein_idx[indiv_id] = 0;
        nb_proteins[indiv_id] = 0;
        nb_promoters[indiv_id] = 0;

        if (indiv_id == 0) {
            nb_mutations[nb_indiv] = 0;

            rna_idx[nb_indiv] = 0;
            protein_idx[nb_indiv] = 0;
            nb_proteins[nb_indiv] = 0;
            nb_promoters[nb_indiv] = 0;
        }
    }
}
